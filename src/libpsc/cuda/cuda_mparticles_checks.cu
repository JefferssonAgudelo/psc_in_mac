#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_bits.h"

// ----------------------------------------------------------------------
// check_in_patch_unordered_slow

template<typename BS>
bool cuda_mparticles<BS>::check_in_patch_unordered_slow()
{
  uint n_prts_by_patch[this->n_patches];
  this->get_size_all(n_prts_by_patch);

  uint off = 0;
  for (int p = 0; p < this->n_patches; p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->d_xi4[off + n], p);
      if (!(bidx >= 0 && bidx <= this->n_blocks)) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_in_patch_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bix_id_unordered_slow
//
// checks that block indices are correct,
// id is just enumerating particles

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_id_unordered_slow()
{
  uint n_prts_by_patch[this->n_patches];
  this->get_size_all(n_prts_by_patch);

  uint off = 0;
  for (int p = 0; p < this->n_patches; p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = this->blockIndex(this->d_xi4[off + n], p);
      if (!(bidx == this->by_block_.d_idx[off+n])) return false;
      if (!(off+n == this->by_block_.d_id[off+n])) return false;
    }
    off += n_prts_by_patch[p];
  }

  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_bidx_id_unordered_slow()\n");
  return true;
}

// ----------------------------------------------------------------------
// check_ordered

template<typename BS>
bool cuda_mparticles<BS>::check_ordered()
{
  thrust::host_vector<float4> h_xi4(this->d_xi4.data(), this->d_xi4.data() + this->n_prts);
  thrust::host_vector<uint> h_off(this->by_block_.d_off);
  thrust::host_vector<uint> h_id(this->by_block_.d_id.data(), this->by_block_.d_id.data() + this->n_prts);

  //printf("check_ordered: need_reorder %s\n", need_reorder ? "true" : "false");

  uint off = 0;
  for (int b = 0; b < this->n_blocks; b++) {
    int p = b / this->n_blocks_per_patch;
    uint off_b = h_off[b], off_e = h_off[b+1];
    if (!(off_e >= off_b)) return false;
    //printf("check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    if (!(off_b == off)) return false;
    for (int n = h_off[b]; n < h_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = h_xi4[h_id[n]];
      } else {
	xi4 = h_xi4[n];
      }
      uint bidx = this->blockIndex(xi4, p);
      //printf("check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("check_ordered: b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	Int3 bpos = this->blockPosition(&xi4.x);
	printf("block_pos %d %d\n", bpos[1], bpos[2]);
      }
      if (!(b == bidx)) return false;
    }
    off += off_e - off_b;
  }
  if (!(off == this->n_prts)) return false;
  // printf("PASS: cuda_mparticles_check_ordered:\n");
  return true;
}

// ----------------------------------------------------------------------
// check_bidx

template<typename BS>
bool cuda_mparticles<BS>::check_bidx_after_push()
{
  bool ok = true;
  
  for (int p = 0; p < this->n_patches; p++) {
    int begin = this->by_block_.d_off[p * this->n_blocks_per_patch];
    int end = this->by_block_.d_off[(p+1) * this->n_blocks_per_patch];
    for (int n = begin; n < end; n++) {
      float4 xi4 = this->d_xi4[n];
      int bidx = this->by_block_.d_idx[n];
      int bidx2 = this->blockIndex(xi4, p);
      if (bidx2 < 0) bidx2 = this->n_blocks;
      if (bidx != bidx2) {
	mprintf("check_bidx: n %d: xi4 %g %g %g bidx %d/%d\n", n, xi4.x, xi4.y, xi4.z,
		bidx, bidx2);
	ok = false;
      }
    }
  }
  return ok;
}

