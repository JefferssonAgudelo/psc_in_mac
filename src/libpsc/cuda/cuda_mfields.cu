#include "hip/hip_runtime.h"

#include "cuda_mfields.h"
#include "cuda_bits.h"

#include "fields.hxx"

#include <cstdio>
#include <cassert>

// ======================================================================
// cuda_mfields

// ----------------------------------------------------------------------
// ctor

cuda_mfields::cuda_mfields(const Grid_t& grid, int _n_fields, const Int3& ibn)
  : ib(-ibn),
    im(grid.ldims + 2 * ibn),
    n_patches(grid.n_patches()),
    n_fields(_n_fields),
    n_cells_per_patch(im[0] * im[1] * im[2]),
    n_cells(n_patches * n_cells_per_patch),
    d_flds_(n_fields * n_cells),
    grid_(grid)
{
  cuda_base_init();
}

// ----------------------------------------------------------------------
// to_json

mrc_json_t cuda_mfields::to_json()
{
  mrc_json_t json = mrc_json_object_new(9);
  mrc_json_object_push_integer(json, "n_patches", n_patches);
  mrc_json_object_push_integer(json, "n_fields", n_fields);
  mrc_json_object_push_integer(json, "n_cells_per_patch", n_cells_per_patch);
  mrc_json_object_push_integer(json, "n_cells", n_cells);

  mrc_json_object_push(json, "ib", mrc_json_integer_array_new(3, ib));
  mrc_json_object_push(json, "im", mrc_json_integer_array_new(3, im));

  mrc_json_t json_flds = mrc_json_object_new(2);
  mrc_json_object_push(json, "flds", json_flds);
  mrc_json_object_push_boolean(json_flds, "__field5d__", true);
  mrc_json_t json_flds_patches = mrc_json_array_new(n_patches);
  mrc_json_object_push(json_flds, "data", json_flds_patches);

  fields_single_t flds = get_host_fields();
  Fields3d<fields_single_t> F(flds);
  for (int p = 0; p < n_patches; p++) {
    copy_from_device(p, flds, 0, n_fields);

    mrc_json_t json_flds_comps = mrc_json_array_new(n_fields);
    mrc_json_array_push(json_flds_patches, json_flds_comps);
    for (int m = 0; m < n_fields; m++) {
      mrc_json_t json_fld_z = mrc_json_array_new(im[2]);
      mrc_json_array_push(json_flds_comps, json_fld_z);
      for (int k = ib[2]; k < ib[2] + im[2]; k++) {
	mrc_json_t json_fld_y = mrc_json_array_new(im[1]);
	mrc_json_array_push(json_fld_z, json_fld_y);
	for (int j = ib[1]; j < ib[1] + im[1]; j++) {
	  mrc_json_t json_fld_x = mrc_json_array_new(im[0]);
	  mrc_json_array_push(json_fld_y, json_fld_x);
	  for (int i = ib[0]; i < ib[0] + im[0]; i++) {
	    mrc_json_array_push_double(json_fld_x, F(m, i,j,k));
	  }
	}
      }
    }
  }
  flds.dtor();

  return json;
}

// ----------------------------------------------------------------------
// dump

void cuda_mfields::dump(const char *filename)
{
  mrc_json_t json = to_json();

  const char *buf = mrc_json_to_string(json);
  if (filename) {
    FILE *file = fopen(filename, "w");
    assert(file);
    fwrite(buf, 1, strlen(buf), file);
    fclose(file);
  } else {
    printf("cuda_mfields (json):\n%s\n", buf);
  }
  free((void *) buf);

  // FIXME free json
}

// ----------------------------------------------------------------------
// cast to DMFields

cuda_mfields::operator DMFields()
{
  return DMFields(d_flds_.data().get(), n_cells_per_patch * n_fields, im, ib);
}

// ----------------------------------------------------------------------
// operator[]

DFields cuda_mfields::operator[](int p)
{
  return static_cast<DMFields>(*this)[p];
}

// ----------------------------------------------------------------------
// get_host_fields

fields_single_t cuda_mfields::get_host_fields()
{
  return fields_single_t(grid(), ib, im, n_fields);
}

// ----------------------------------------------------------------------
// copy_to_device

void cuda_mfields::copy_to_device(int p, fields_single_t h_flds, int mb, int me)
{
  hipError_t ierr;
  
  if (mb == me) {
    return;
  }
  assert(mb < me);

  uint size = n_cells_per_patch;
  ierr = hipMemcpy((*this)[p].data() + mb * size,
		    h_flds.data() + mb * size,
		    (me - mb) * size * sizeof(float),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// copy_from_device

void cuda_mfields::copy_from_device(int p, fields_single_t h_flds, int mb, int me)
{
  hipError_t ierr;

  if (mb == me) {
    return;
  }
  assert(mb < me);

  uint size = n_cells_per_patch;
  ierr = hipMemcpy(h_flds.data() + mb * size,
		    (*this)[p].data() + mb * size,
		    (me - mb) * size * sizeof(float),
		    hipMemcpyDeviceToHost); cudaCheck(ierr);
}

#define BND (2) // FIXME
#define X3_DEV_OFF_YZ(fldnr, jy,jz)					\
  ((((fldnr)								\
     *mz + ((jz)+2))							\
    *my + ((jy)+2))							\
   *1 + (0))

#define F3_DDEV(d_flds, fldnr,ix,jy,jz)		\
  (d_flds)[X3_DEV_OFF_YZ(fldnr, jy,jz)]

#define BLOCKSIZE_Y 16
#define BLOCKSIZE_Z 16

// ----------------------------------------------------------------------
// axpy_comp_yz

__global__ static void
k_axpy_comp_yz(float *y_flds, int ym, float a, float *x_flds, int xm,
	     int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy >= my || iz >= mz) {
    return;
  }

  iy -= BND;
  iz -= BND;

  F3_DDEV(y_flds, ym, 0,iy,iz) += a * F3_DDEV(x_flds, xm, 0,iy,iz);
}

void cuda_mfields::axpy_comp_yz(int ym, float a, cuda_mfields *cmflds_x, int xm)
{
  int my = im[1];
  int mz = im[2];
  assert(ib[1] == -BND);
  assert(ib[2] == -BND);

  dim3 dimGrid((my + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
	       (mz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);

  for (int p = 0; p < n_patches; p++) {
    k_axpy_comp_yz<<<dimGrid, dimBlock>>>((*this)[p].data(), ym, a,
					  (*cmflds_x)[p].data(), xm, my, mz);
  }
  cuda_sync_if_enabled();
}

// ----------------------------------------------------------------------
// zero_comp_yz

// FIXME, this should be more easily doable by just a hipMemset()

__global__ static void
k_zero_comp_yz(float *x_flds, int xm, int my, int mz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy >= my || iz >= mz) {
    return;
  }

  iy -= BND;
  iz -= BND;

  F3_DDEV(x_flds, xm, 0,iy,iz) = 0.f;
}

__global__ static void
k_zero_comp_xyz(float *data, uint n, uint stride)
{
  uint i = blockIdx.x * blockDim.x + threadIdx.x;
  uint p = blockIdx.y;

  if (i < n) {
    data[i + p * stride] = 0.f;
  }
}

void cuda_mfields::zero_comp(int m, dim_yz tag)
{
  int my = im[1];
  int mz = im[2];
  assert(ib[1] == -BND);
  assert(ib[2] == -BND);

  dim3 dimGrid((my + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
	       (mz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);

  // OPT, should be done in a single kernel
  for (int p = 0; p < n_patches; p++) {
    k_zero_comp_yz<<<dimGrid, dimBlock>>>((*this)[p].data(), m, my, mz);
  }
  cuda_sync_if_enabled();
}

void cuda_mfields::zero_comp(int m, dim_xyz tag)
{
  int n = n_cells_per_patch;
  int stride = n * n_fields;

  const int THREADS_PER_BLOCK = 512;
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, n_patches);

  k_zero_comp_xyz<<<dimGrid, dimBlock>>>(data() + m * n, n, stride);
  cuda_sync_if_enabled();
}

