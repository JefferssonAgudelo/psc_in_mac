#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_bndp.h"
#include "cuda_test.hxx"

#include <mrc_profile.h>

#include "gtest/gtest.h"

struct prof_globals prof_globals; // FIXME

int
prof_register(const char *name, float simd, int flops, int bytes)
{
  return 0;
}

using CudaMparticles = cuda_mparticles<BS444>;

// ======================================================================
// CudaMparticlesBndTest

struct CudaMparticlesBndTest : TestBase<CudaMparticles>, ::testing::Test
{
  using Double3 = Vec3<double>;
  
  std::unique_ptr<Grid_t> grid;
  std::unique_ptr<CudaMparticles> cmprts;
  std::unique_ptr<cuda_bndp<CudaMparticles, dim_xyz>> cbndp;

  void SetUp()
  {
    auto domain = Grid_t::Domain{{32, 32, 32}, {320., 320., 320.}, {0., 0., 0.},
				 {2, 2, 2}};
    auto bc = GridBc{};
    auto kinds = Grid_t::Kinds{Grid_t::Kind{1., 1., "k0"},
			       Grid_t::Kind{1., 1., "k1"},
			       Grid_t::Kind{1., 1., "k2"},
			       Grid_t::Kind{1., 1., "k3"}};
    auto norm = Grid_t::Normalization{};
    double dt = .1;
    grid.reset(new Grid_t(domain, bc, kinds, norm, dt));

    grid->kinds.push_back(Grid_t::Kind(1., 1., "test species"));

    cmprts.reset(make_cmprts(*grid));

    // (ab)use kind to track particle more easily in the test
    std::vector<cuda_mparticles_prt> prts = {
      {{.5,  35., 5.}, {}, 0., 0},
      {{.5, 155., 5.}, {}, 0., 1},
      
      {{.5,  35., 5.}, {}, 0., 2},
      {{.5, 155., 5.}, {}, 0., 3},
    };

    std::vector<uint> n_prts_by_patch = {2, 2, 0, 0, 0, 0, 0, 0};
    
    // FIXME eventually shouldn't have to reserve additional room for sending here
    std::vector<uint> n_prts_reserve_by_patch = {2, 4, 0, 0, 0, 0, 0, 0};
    
    cmprts->reserve_all(n_prts_reserve_by_patch.data());
    cmprts->inject_buf(prts.data(), n_prts_by_patch.data());

    // move every particle one full cell to the right (+y, that is)
    // (position doesn't actually matter since we'll only look at bidx)
    for (int n = 0; n < cmprts->n_prts; n++) {
      float4 xi4 = cmprts->d_xi4[n];
      xi4.y += 10.;
      cmprts->d_xi4[n] = xi4;
    }
    auto& d_bidx = cmprts->by_block_.d_idx;
    d_bidx[0] = 4;
    d_bidx[1] = cmprts->n_blocks + 0; // oob p0
    d_bidx[2] = 68;
    d_bidx[3] = cmprts->n_blocks + 1; // oob p1
    
#if 0
    cmprts->dump();
#endif

    cbndp.reset(new cuda_bndp<CudaMparticles, dim_xyz>(*grid));
  }
};


// ----------------------------------------------------------------------
// BndPrep
//
// tests cuda_bndp::prep()

TEST_F(CudaMparticlesBndTest, BndPrep)
{
  cbndp->prep(cmprts.get());

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bpatch[0].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[1].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[0].buf[0].kind, 1);
  EXPECT_EQ(cbndp->bpatch[1].buf[0].kind, 3);
}

// ----------------------------------------------------------------------
// BndPrepDetail
//
// tests the pieces that go into cuda_bndp::prep()

struct is_inside
{
  is_inside(int n_blocks) : n_blocks_(n_blocks) {}
  
  __host__ __device__
  bool operator()(thrust::tuple<uint, float4, float4> tup)
  {
    uint bidx = thrust::get<0>(tup);
    return bidx < n_blocks_;
  }
  
  int n_blocks_;
};

TEST_F(CudaMparticlesBndTest, BndPrepDetail)
{
  auto& cmprts = *this->cmprts;

  auto& d_bidx = cmprts.by_block_.d_idx;
#if 0
  for (int n = 0; n < cmprts.n_prts; n++) {
    float4 xi4 = cmprts.d_xi4[n];
    printf("n %d: %g:%g:%g kind %d bidx %d\n", n, xi4.x, xi4.y, xi4.z,
	   cuda_float_as_int(xi4.w), int(d_bidx[n]));
  }
#endif

  auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_bidx.begin(), cmprts.d_xi4.begin(), cmprts.d_pxi4.begin()));
  auto end = thrust::make_zip_iterator(thrust::make_tuple(d_bidx.end(), cmprts.d_xi4.end(), cmprts.d_pxi4.end()));
  auto oob = thrust::stable_partition(begin, end, is_inside(cmprts.n_blocks));

#if 0
  for (int n = 0; n < cmprts.n_prts; n++) {
    float4 xi4 = cmprts.d_xi4[n];
    printf("n %d: %g:%g:%g kind %d bidx %d\n", n, xi4.x, xi4.y, xi4.z,
	   cuda_float_as_int(xi4.w), int(d_bidx[n]));
  }
#endif

  EXPECT_EQ(oob, begin + 2);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[0]).w), 0);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[1]).w), 2);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[2]).w), 1);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[3]).w), 3);

  cbndp->n_prts_send = end - oob;

  EXPECT_EQ(cmprts.n_prts, 4);
  EXPECT_EQ(cbndp->n_prts_send, 2);

  cmprts.n_prts -= cbndp->n_prts_send;

  // particles 1, 3, which need to be exchanged, should now be at the
  // end of the regular array
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[cmprts.n_prts  ]).w), 1);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[cmprts.n_prts+1]).w), 3);

  // test copy_from_dev_and_convert
  cbndp->copy_from_dev_and_convert(&cmprts, cbndp->n_prts_send);

#if 0
  for (int p = 0; p < cmprts.n_patches; p++) {
    printf("from_dev: p %d\n", p);
    for (auto& prt : cbndp->bpatch[p].buf) {
      printf("  prt xyz %g %g %g kind %d\n", prt.xi, prt.yi, prt.zi, prt.kind_);
    }
  }
#endif

  EXPECT_EQ(cbndp->bpatch[0].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[1].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[0].buf[0].kind, 1);
  EXPECT_EQ(cbndp->bpatch[1].buf[0].kind, 3);
}

// ----------------------------------------------------------------------
// BndPost
//
// tests cuda_bndp::post()

TEST_F(CudaMparticlesBndTest, BndPost)
{
  auto& cmprts = *this->cmprts;
  // BndPost expects the work done by bnd_prep()
  cbndp->prep(&cmprts);

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bpatch[0].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[1].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[0].buf[0].kind, 1);
  EXPECT_EQ(cbndp->bpatch[1].buf[0].kind, 3);

  // Mock what the actual boundary exchange does, ie., move
  // particles to their new patch and adjust the relative position.
  // This assumes periodic b.c.
  particle_cuda_t prt1 = cbndp->bpatch[0].buf[0];
  particle_cuda_t prt3 = cbndp->bpatch[1].buf[0];
  prt1.x[1] -= 40.;
  prt3.x[1] -= 40.;
  cbndp->bpatch[0].buf[0] = prt3;
  cbndp->bpatch[1].buf[0] = prt1;
  
  cbndp->post(&cmprts);

  // bnd_post doesn't do the actual final reordering
  EXPECT_TRUE(cmprts.need_reorder);
  cmprts.reorder();
  EXPECT_TRUE(cmprts.check_ordered());

#if 0
  cmprts.dump();
#endif
}

// ----------------------------------------------------------------------
// BndPostDetail
//
// tests the pieces that go into cuda_bndp::post()

TEST_F(CudaMparticlesBndTest, BndPostDetail)
{
  auto& cmprts = *this->cmprts;
  // BndPost expects the work done by bnd_prep()
  cbndp->prep(&cmprts);

  // particles 0 and 2 remain in their patch,
  // particles 1 and 3 leave their patch and need special handling
  EXPECT_EQ(cbndp->bpatch[0].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[1].buf.size(), 1);
  EXPECT_EQ(cbndp->bpatch[0].buf[0].kind, 1);
  EXPECT_EQ(cbndp->bpatch[1].buf[0].kind, 3);

  // Mock what the actual boundary exchange does, ie., move
  // particles to their new patch and adjust the relative position.
  // This assumes periodic b.c.
  particle_cuda_t prt1 = cbndp->bpatch[0].buf[0];
  particle_cuda_t prt3 = cbndp->bpatch[1].buf[0];
  prt1.x[1] -= 160.;
  prt3.x[1] -= 160.;
  cbndp->bpatch[0].buf[0] = prt3;
  cbndp->bpatch[1].buf[0] = prt1;

  // === test convert_and_copy_to_dev()
  uint n_prts_recv = cbndp->convert_and_copy_to_dev(&cmprts);
  cmprts.n_prts += n_prts_recv;

  // n_recv should be set for each patch, and its total
  EXPECT_EQ(cbndp->bpatch[0].n_recv, 1);
  EXPECT_EQ(cbndp->bpatch[1].n_recv, 1);
  EXPECT_EQ(n_prts_recv, 2);

  // the received particle have been appended to the two remaining ones
  EXPECT_EQ(cmprts.n_prts, 4);

  // and the particle have been appended after the old end of the particle list
  int n_prts_old = cmprts.n_prts - n_prts_recv;
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[n_prts_old  ]).w), 3);
  EXPECT_EQ(cuda_float_as_int(float4(cmprts.d_xi4[n_prts_old+1]).w), 1);

  // block indices have been calculated
  auto& d_bidx = cmprts.by_block_.d_idx;
  EXPECT_EQ(d_bidx[n_prts_old  ], 0);  // 0th block in 0th patch
  EXPECT_EQ(d_bidx[n_prts_old+1], 64); // 0th block in 1st patch

  cmprts.resize(cmprts.n_prts);
  thrust::sequence(cmprts.by_block_.d_id.begin(), cmprts.by_block_.d_id.end());
  thrust::stable_sort_by_key(d_bidx.begin(), d_bidx.end(), cmprts.by_block_.d_id.begin());

#if 0
  for (int n = 0; n < cmprts.n_prts; n++) {
    float4 xi4 = cmprts.d_xi4[n];
    printf("n %d: bidx %d id %d\n", n,
	   int(d_bidx[n]), int(cmprts.by_block_.d_id[n]));
  }
#endif
  
  EXPECT_EQ(cmprts.n_prts, 4);
  auto& d_id = cmprts.by_block_.d_id;
  EXPECT_EQ(d_id[0], 2);
  EXPECT_EQ(d_id[1], 0);
  EXPECT_EQ(d_id[2], 3);
  EXPECT_EQ(d_id[3], 1);

  // find offsets
  thrust::counting_iterator<uint> search_begin(0);
  thrust::upper_bound(d_bidx.begin(), d_bidx.end(),
		      search_begin, search_begin + cmprts.n_blocks,
		      cmprts.by_block_.d_off.begin() + 1);
  // d_off[0] was set to zero during d_off initialization
  auto& d_off = cmprts.by_block_.d_off;
  for (int b = 0; b <= cmprts.n_blocks; b++) {
    //if (b < cmprts.n_blocks) printf("b %d: off [%d:%d[\n", b, int(d_off[b]), int(d_off[b+1]));
    if (b < 1) {
      EXPECT_EQ(d_off[b], 0) << "where b = " << b;
    } else if (b < 5) {
      EXPECT_EQ(d_off[b], 1) << "where b = " << b;
    } else if (b < 65) {
      EXPECT_EQ(d_off[b], 2) << "where b = " << b;
    } else if (b < 69) {
      EXPECT_EQ(d_off[b], 3) << "where b = " << b;
    } else {
      EXPECT_EQ(d_off[b], 4) << "where b = " << b;
    }
  }

  cmprts.need_reorder = true;

  // bnd_post doesn't do the actually final reordering, but
  // let's do it here for a final check
  cmprts.reorder();
  // for (int n = 0; n < cmprts.n_prts; n++) {
  //   float4 xi4 = cmprts.d_xi4[n];
  //   printf("n %d: %g:%g kind %d\n", n, xi4.y, xi4.z, cuda_float_as_int(xi4.w));
  // }
  EXPECT_TRUE(cmprts.check_ordered());

#if 0
  cmprts.dump();
#endif
}

// ======================================================================
// main

int main(int argc, char **argv)
{
  MPI_Init(&argc, &argv);

  ::testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();

  MPI_Finalize();
  return rc;
}
